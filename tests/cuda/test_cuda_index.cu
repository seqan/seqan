// ==========================================================================
//                 SeqAn - The Library for Sequence Analysis
// ==========================================================================
// Copyright (c) 2006-2013, Knut Reinert, FU Berlin
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
//     * Redistributions of source code must retain the above copyright
//       notice, this list of conditions and the following disclaimer.
//     * Redistributions in binary form must reproduce the above copyright
//       notice, this list of conditions and the following disclaimer in the
//       documentation and/or other materials provided with the distribution.
//     * Neither the name of Knut Reinert or the FU Berlin nor the names of
//       its contributors may be used to endorse or promote products derived
//       from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL KNUT REINERT OR THE FU BERLIN BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
// LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY
// OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
// DAMAGE.
//
// ==========================================================================
// Author: Enrico Siragusa <enrico.siragusa@fu-berlin.de>
// ==========================================================================

#include <seqan/basic.h>
#include <seqan/sequence.h>
#include <seqan/index.h>
#include <seqan/index/index_fm_device.h>

#include "test_cuda_common.h"

using namespace seqan;

// TODO(esiragusa): move this into metaprogramming algebra
namespace seqan {

template <typename T1, typename T2>
struct Pair<T1, T2, Tag<void> > {};

// Manually specialize word size to be compatible with GPU

template <typename TValue>
struct RankDictionaryWordSize_<TValue, TwoLevels<void> > :
    BitsPerValue<__uint32> {};

template <typename TValue>
struct RankDictionaryWordSize_<TValue, TwoLevels<Device<void> > > :
    BitsPerValue<__uint32> {};

template <typename TValue, typename TSpec>
struct RankDictionaryWordSize_<TValue, TwoLevels<View<TSpec> > > :
    BitsPerValue<__uint32> {};

}

// ============================================================================
// Types
// ============================================================================

typedef TagList<FibreRawText,
        TagList<FibreLF
        > >
    FMIndexFibres;

// ============================================================================
// Classes
// ============================================================================

// ----------------------------------------------------------------------------
// Class CudaIndexTest
// ----------------------------------------------------------------------------

template <typename TType>
class CudaIndexTest : public Test
{
public:
    typedef TType                                       TIndex;
    typedef typename Host<TIndex>::Type                 TText;
    typedef typename Device<TIndex>::Type               TCudaIndex;

    TText    text;
    TIndex   index;

    CudaIndexTest() :
        text(),
        index(text)
    {
        // TODO(esiragusa): init generic text.
        appendValue(text, "ACGTACGTACGTACGTACGTACGTACGTACGTACGTACGTACGTACGTACGTACGTACGTACGTACGTACGTACGTACGT");

        // TODO(esiragusa): reverse text on FM-index only.
        reverse(text);
        indexCreate(index);
        reverse(text);
    }
};

typedef TagList<DnaStringSetFMIndex> CudaIndexTestTypes;

SEQAN_TYPED_TEST_CASE(CudaIndexTest, CudaIndexTestTypes);

// ----------------------------------------------------------------------------
// Class CudaIndexFibreTest
// ----------------------------------------------------------------------------

template <typename TTypes>
class CudaIndexFibreTest : public CudaIndexTest<typename Value<TTypes, 1>::Type> {};

// TODO(esiragusa): use metaprogramming algebra.
//typedef Product<DnaStringSetFMIndex, FMIndexFibres>::Type CudaIndexFibreTestTypes;

typedef TagList<Pair<DnaStringSetFMIndex, FibreRawText, Tag<void> >,
        TagList<Pair<DnaStringSetFMIndex, FibreLF, Tag<void> >
        > >
    CudaIndexFibreTestTypes;

SEQAN_TYPED_TEST_CASE(CudaIndexFibreTest, CudaIndexFibreTestTypes);

// ----------------------------------------------------------------------------
// Class CudaIndexCountTest
// ----------------------------------------------------------------------------

template <typename TTypes>
class CudaIndexCountTest : public CudaIndexTest<typename Value<TTypes, 1>::Type>
{
public:
    typedef typename Value<TTypes, 1>::Type             TIndex;
    typedef typename Value<TTypes, 2>::Type             TNeedles;
    typedef CudaIndexTest<TIndex>                       TBase;
    typedef typename Size<TIndex>::Type                 TSize;
    typedef typename Device<TNeedles>::Type             TCudaNeedles;

    TNeedles needles;
    TSize    occurrences;

    CudaIndexCountTest() :
        TBase()
    {
        // TODO(esiragusa): append generic needles.
        appendValue(needles, "ACGT");
        appendValue(needles, "CGT");
        appendValue(needles, "GTA");

        occurrences = countOccurrences(this->index, needles);
    }
};

// TODO(esiragusa): use metaprogramming algebra.
//typedef Product<DnaStringSetFMIndex, DnaStringSet>::Type CudaIndexCountTestTypes;

typedef TagList<Pair<DnaStringSetFMIndex, DnaStringSet, Tag<void> > > CudaIndexCountTestTypes;

SEQAN_TYPED_TEST_CASE(CudaIndexCountTest, CudaIndexCountTestTypes);

// ============================================================================
// Tests
// ============================================================================

// ----------------------------------------------------------------------------
// Test assign()
// ----------------------------------------------------------------------------

SEQAN_TYPED_TEST(CudaIndexTest, Assign)
{
    typedef typename TestFixture::TIndex        TIndex;
    typedef typename TestFixture::TCudaIndex    TCudaIndex;

    hipDeviceReset();

    TCudaIndex cudaIndex;
    assign(cudaIndex, this->index);
    SEQAN_ASSERT_EQ(length(cudaIndex), length(this->index));

//    TIndex        index;
//    assign(index, cudaIndex);
//    SEQAN_ASSERT(index == this->index);
}

// ----------------------------------------------------------------------------
// Test value() on Index Fibres
// ----------------------------------------------------------------------------

SEQAN_TYPED_TEST(CudaIndexFibreTest, Values)
{
    typedef FibreLF                                 TTag;
    typedef typename TestFixture::TIndex            TIndex;
    typedef typename TestFixture::TCudaIndex        TCudaIndex;
    typedef typename Fibre<TIndex, TTag>::Type      TFibre;
    typedef typename Fibre<TCudaIndex, TTag>::Type  TCudaFibre;
    typedef typename View<TCudaFibre>::Type         TCudaFibreView;
    typedef typename Size<TFibre>::Type             TSize;

    hipDeviceReset();

    TCudaIndex cudaIndex;
    assign(cudaIndex, this->index);

    TFibre & fibre = getFibre(this->index, TTag());
    TCudaFibre & cudaFibre = getFibre(cudaIndex, TTag());
    SEQAN_ASSERT_EQ(length(fibre), length(cudaFibre));

    TCudaFibreView cudaFibreView = view(cudaFibre);
    for (TSize pos = 0; pos < length(fibre); pos++)
    {
        testGetValue<<<1,1>>>(cudaFibreView, pos, fibre[pos]);
        hipDeviceSynchronize();
        SEQAN_ASSERT_EQ(hipGetLastError(), hipSuccess);
    }
}

// ----------------------------------------------------------------------------
// Test countOccurrences()
// ----------------------------------------------------------------------------

SEQAN_TYPED_TEST(CudaIndexCountTest, Count)
{
    typedef typename TestFixture::TCudaIndex    TCudaIndex;
    typedef typename TestFixture::TCudaNeedles  TCudaNeedles;

    hipDeviceReset();

    TCudaIndex   cudaIndex;
    TCudaNeedles cudaNeedles;

    assign(cudaIndex, this->index);
    assign(cudaNeedles, this->needles);

    SEQAN_ASSERT_EQ(countOccurrences(cudaIndex, cudaNeedles), this->occurrences);
}

// ============================================================================
// Register Tests
// ============================================================================

int main(int argc, char const ** argv)
{
    TestSystem::init(argc, argv);
    return TestSystem::runAll();
}
