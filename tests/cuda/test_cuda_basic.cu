// ==========================================================================
//                 SeqAn - The Library for Sequence Analysis
// ==========================================================================
// Copyright (c) 2006-2013, Knut Reinert, FU Berlin
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
//     * Redistributions of source code must retain the above copyright
//       notice, this list of conditions and the following disclaimer.
//     * Redistributions in binary form must reproduce the above copyright
//       notice, this list of conditions and the following disclaimer in the
//       documentation and/or other materials provided with the distribution.
//     * Neither the name of Knut Reinert or the FU Berlin nor the names of
//       its contributors may be used to endorse or promote products derived
//       from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL KNUT REINERT OR THE FU BERLIN BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
// LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY
// OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
// DAMAGE.
//
// ==========================================================================
// Author: Enrico Siragusa <enrico.siragusa@fu-berlin.de>
// ==========================================================================

#include <seqan/basic.h>

using namespace seqan;

// ============================================================================
// Tests
// ============================================================================

// ----------------------------------------------------------------------------
// Test test_cuda_arch
// ----------------------------------------------------------------------------

SEQAN_GLOBAL void testCudaArch()
{
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 200
#error CUDA architecture 2.0 or higher is required.
#endif
}

SEQAN_DEFINE_TEST(test_cuda_arch)
{
    testCudaArch<<<1,1>>>();
    hipDeviceSynchronize();
    SEQAN_ASSERT_EQ(hipGetLastError(), hipSuccess);
}

// ----------------------------------------------------------------------------
// Test test_cuda_assert
// ----------------------------------------------------------------------------

SEQAN_GLOBAL void testCudaAssert()
{
//    asm("trap;");
    SEQAN_ASSERT(false);
}

SEQAN_DEFINE_TEST(test_cuda_assert)
{
    testCudaAssert<<<1,1>>>();
    hipDeviceSynchronize();
    SEQAN_ASSERT_NEQ(hipGetLastError(), hipSuccess);
}

// ============================================================================
// Register Tests
// ============================================================================

SEQAN_BEGIN_TESTSUITE(test_cuda_basic)
{
    // Call tests.
    SEQAN_CALL_TEST(test_cuda_arch);
    SEQAN_CALL_TEST(test_cuda_assert);
}
SEQAN_END_TESTSUITE
