// ==========================================================================
//                 SeqAn - The Library for Sequence Analysis
// ==========================================================================
// Copyright (c) 2006-2013, Knut Reinert, FU Berlin
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
//     * Redistributions of source code must retain the above copyright
//       notice, this list of conditions and the following disclaimer.
//     * Redistributions in binary form must reproduce the above copyright
//       notice, this list of conditions and the following disclaimer in the
//       documentation and/or other materials provided with the distribution.
//     * Neither the name of Knut Reinert or the FU Berlin nor the names of
//       its contributors may be used to endorse or promote products derived
//       from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL KNUT REINERT OR THE FU BERLIN BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
// LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY
// OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
// DAMAGE.
//
// ==========================================================================
// Author: Enrico Siragusa <enrico.siragusa@fu-berlin.de>
// ==========================================================================

#include <seqan/basic.h>
#include <seqan/sequence.h>
#include <seqan/stream.h>

#include <thrust/count.h>

using namespace seqan;

int main(int argc, char const ** argv)
{
    typedef String<char, MMap<> >               TMMapString;
    typedef typename Device<TMMapString>::Type  TDeviceString;

    if (argc != 3)
    {
        std::cerr << "USAGE: " << argv[0] << " <FILENAME> <CHAR>" << std::endl;
        return 1;
    }

    TMMapString str;

    if (!open(str, argv[1], OPEN_RDWR | OPEN_APPEND))
        return 1;

    TDeviceString deviceStr;
    assign(deviceStr, str);

    close(str);

    std::cout << thrust::count(begin(deviceStr, Standard()), end(deviceStr, Standard()), argv[2][0]) << std::endl;

    return 0;
}
