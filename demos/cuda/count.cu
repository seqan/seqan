// ==========================================================================
//                 SeqAn - The Library for Sequence Analysis
// ==========================================================================
// Copyright (c) 2013 NVIDIA Corporation
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
//     * Redistributions of source code must retain the above copyright
//       notice, this list of conditions and the following disclaimer.
//     * Redistributions in binary form must reproduce the above copyright
//       notice, this list of conditions and the following disclaimer in the
//       documentation and/or other materials provided with the distribution.
//     * Neither the name of NVIDIA Corporation nor the names of
//       its contributors may be used to endorse or promote products derived
//       from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
// LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY
// OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
// DAMAGE.
//
// ==========================================================================
// Author: Enrico Siragusa <enrico.siragusa@fu-berlin.de>
// ==========================================================================

#ifndef SEQAN_DEMOS_CUDA_COUNT_H
#define SEQAN_DEMOS_CUDA_COUNT_H

#include <seqan/basic.h>
#include <seqan/sequence.h>
#include <seqan/index.h>
#include <seqan/index/index_fm_device.h>

using namespace seqan;

// ==========================================================================
// Metafunctions
// ==========================================================================

namespace seqan {

// Manually specialize word size to be compatible with GPU

template <typename TValue>
struct RankDictionaryWordSize_<TValue, TwoLevels<void> > :
    BitsPerValue<__uint32> {};

template <typename TValue>
struct RankDictionaryWordSize_<TValue, TwoLevels<Device<void> > > :
    BitsPerValue<__uint32> {};

template <typename TValue, typename TSpec>
struct RankDictionaryWordSize_<TValue, TwoLevels<View<TSpec> > > :
    BitsPerValue<__uint32> {};

}

// ==========================================================================
// Functions
// ==========================================================================

// --------------------------------------------------------------------------
// Function main()
// --------------------------------------------------------------------------

int main(int argc, char const ** argv)
{
    // ----------------------------------------------------------------------
    // Parse input data.
    // ----------------------------------------------------------------------

    // Select the input types.
    typedef DnaString                                       THaystack;
    typedef StringSet<DnaString, Owner<ConcatDirect<> > >   TNeedles;

    if (argc < 3)
    {
        std::cerr << "USAGE: " << argv[0] << " <TEXT> <PATTERN> [<PATTERN> ...]" << std::endl;
        return 1;
    }

    // Create a haystack.
    THaystack haystack = argv[1];

    // Create a set of needles.
    TNeedles needles;
    for (int i = 2; i < argc; i++)
        appendValue(needles, argv[i]);

    // ----------------------------------------------------------------------
    // Build the FM-index on the CPU.
    // ----------------------------------------------------------------------

    // Select the index type.
    typedef Index<THaystack, FMIndex<void, CudaFMIndexConfig> > TIndex;

    // Build the index over the reversed haystack.
    TIndex index(haystack);
    reverse(haystack);
    indexCreate(index);
    reverse(haystack);

    // ----------------------------------------------------------------------
    // Count on the CPU.
    // ----------------------------------------------------------------------

    omp_set_num_threads(8);
    std::cout << "CPU Occurrences: " << countOccurrences(index, needles) << std::endl;

    // ----------------------------------------------------------------------
    // Copy data to the GPU.
    // ----------------------------------------------------------------------

    // Select the GPU types.
    typedef Device<TNeedles>::Type     TDeviceNeedles;
    typedef Device<TIndex>::Type       TDeviceIndex;

    // Copy the needles to the GPU.
    TDeviceNeedles deviceNeedles;
    assign(deviceNeedles, needles);

    // Copy the index to the GPU.
    TDeviceIndex deviceIndex;
    assign(deviceIndex, index);

    // ----------------------------------------------------------------------
    // Count on the GPU.
    // ----------------------------------------------------------------------

    std::cout << "GPU Occurrences: " << countOccurrences(deviceIndex, deviceNeedles) << std::endl;

    return 0;
}

#endif  // SEQAN_DEMOS_CUDA_COUNT_H
