// ==========================================================================
//                 SeqAn - The Library for Sequence Analysis
// ==========================================================================
// Copyright (c) 2006-2013, Knut Reinert, FU Berlin
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
//     * Redistributions of source code must retain the above copyright
//       notice, this list of conditions and the following disclaimer.
//     * Redistributions in binary form must reproduce the above copyright
//       notice, this list of conditions and the following disclaimer in the
//       documentation and/or other materials provided with the distribution.
//     * Neither the name of Knut Reinert or the FU Berlin nor the names of
//       its contributors may be used to endorse or promote products derived
//       from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL KNUT REINERT OR THE FU BERLIN BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
// LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY
// OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
// DAMAGE.
//
// ==========================================================================
// Author: Enrico Siragusa <enrico.siragusa@fu-berlin.de>
// ==========================================================================

#include <seqan/basic.h>
#include <seqan/sequence.h>
#include <thrust/count.h>

#include "test_cuda_common.h"

using namespace seqan;

// ============================================================================
// Types
// ============================================================================

typedef TagList<String<char, Alloc<> >,
        TagList<String<Dna, Alloc<> >,
        TagList<String<Dna5, Alloc<> >
//        TagList<String<Dna, Packed<> >
        > > > //>
        StringTypes;

// TODO(esiragusa): test StringSets.
//typedef TagList<StringSet<CharString, Owner<ConcatDirect<> > >,
//        TagList<StringSet<DnaString, Owner<ConcatDirect<> > >
//        > >
//    TStringSetTypes;

// TODO(esiragusa): use metaprogramming algebra.
//typedef Product<StringTypes, Owner<ConcatDirect<> > >::Type TStringSetTypes;

// ============================================================================
// Classes
// ============================================================================

// ----------------------------------------------------------------------------
// Class CudaSequenceTest
// ----------------------------------------------------------------------------

template <typename TType>
class CudaSequenceTest : public Test
{
public:
    typedef TType                             TString;
    typedef typename Device<TString>::Type    TCudaString;
    typedef typename Value<TString>::Type     TAlphabet;

    TString str;

    CudaSequenceTest() :
        str("ACGTACGTACGTACGTACGTACGTACGTACGTACGTACGT")
    {}
};

SEQAN_TYPED_TEST_CASE(CudaSequenceTest, StringTypes);


// ============================================================================
// Tests
// ============================================================================

// ----------------------------------------------------------------------------
// Test assign()
// ----------------------------------------------------------------------------

SEQAN_TYPED_TEST(CudaSequenceTest, Assign)
{
    typedef typename TestFixture::TString       TString;
    typedef typename TestFixture::TCudaString   TCudaString;
    typedef typename TestFixture::TAlphabet     TAlphabet;

    hipDeviceReset();

    TCudaString cudaStr;
    assign(cudaStr, this->str);
    SEQAN_ASSERT_EQ(length(cudaStr), length(this->str));
    SEQAN_ASSERT_EQ(thrust::count(begin(cudaStr, Standard()), end(cudaStr, Standard()), TAlphabet('A')), 10u);
    SEQAN_ASSERT_EQ(thrust::count(begin(cudaStr, Standard()), end(cudaStr, Standard()), TAlphabet('C')), 10u);
    SEQAN_ASSERT_EQ(thrust::count(begin(cudaStr, Standard()), end(cudaStr, Standard()), TAlphabet('G')), 10u);
    SEQAN_ASSERT_EQ(thrust::count(begin(cudaStr, Standard()), end(cudaStr, Standard()), TAlphabet('T')), 10u);

//    TString str;
//    assign(cudaStr, str);
//    SEQAN_ASSERT_EQ(str, this->str);
}

// ----------------------------------------------------------------------------
// Test getValue()
// ----------------------------------------------------------------------------

SEQAN_TYPED_TEST(CudaSequenceTest, GetValue)
{
    typedef typename TestFixture::TString       TString;
    typedef typename TestFixture::TCudaString   TCudaString;
    typedef typename View<TCudaString>::Type    TCudaStringView;
    typedef typename Size<TString>::Type        TSize;

    hipDeviceReset();

    TCudaString cudaStr;
    assign(cudaStr, this->str);
    TCudaStringView cudaStrView = view(cudaStr);

    for (TSize pos = 0; pos < length(this->str); pos++)
    {
        testGetValue<<<1,1>>>(cudaStrView, pos, getValue(this->str, pos));
        hipDeviceSynchronize();
        SEQAN_ASSERT_EQ(hipGetLastError(), hipSuccess);
    }
}

// ============================================================================
// Register Tests
// ============================================================================

int main(int argc, char const ** argv)
{
    TestSystem::init(argc, argv);
    return TestSystem::runAll();
}
